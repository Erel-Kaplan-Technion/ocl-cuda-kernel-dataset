#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <float.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

#include "CudaMath.h"
#include "dds.h"
#include "permutations.h"

// Definitions
#define INPUT_IMAGE     "teapot512_std.ppm"
#define REFERENCE_IMAGE "teapot512_ref.dds"

#define ERROR_THRESHOLD 0.02f

#define NUM_THREADS 64

#define __debugsync()

template <class T> __device__ inline void swap(T &a, T &b)
{
    T tmp = a;
    a     = b;
    b     = tmp;
}

__constant__ float3 kColorMetric = {1.0f, 1.0f, 1.0f};

__device__ void sortColors(const float *values, int *ranks, cg::thread_group tile)
{
    const int tid = threadIdx.x;

    int rank = 0;

#pragma unroll

    for (int i = 0; i < 16; i++) {
        rank += (values[i] < values[tid]);
    }

    ranks[tid] = rank;

    cg::sync(tile);

    for (int i = 0; i < 15; i++) {
        if (tid > i && ranks[tid] == ranks[i]) {
            ++ranks[tid];
        }
        cg::sync(tile);
    }
}

__device__ void loadColorBlock(const uint      *image,
                               float3           colors[16],
                               float3           sums[16],
                               int              xrefs[16],
                               int              blockOffset,
                               cg::thread_block cta)
{
    const int bid = blockIdx.x + blockOffset;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    float3 tmp;

    cg::thread_group tile = cg::tiled_partition(cta, 16);

    if (idx < 16) {
        uint c = image[(bid) * 16 + idx];

        colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

        cg::sync(tile);
        colorSums(colors, sums, tile);

        cg::sync(tile);

        float3 axis = bestFitLine(colors, sums[0], tile);

        cg::sync(tile);

        dps[idx] = dot(colors[idx], axis);

        cg::sync(tile);

        sortColors(dps, xrefs, tile);

        cg::sync(tile);

        tmp = colors[idx];

        cg::sync(tile);

        colors[xrefs[idx]] = tmp;
    }
}

inline __device__ float3 roundAndExpand(float3 v, ushort *w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);

    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f;
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}

__constant__ float     alphaTable4[4] = {9.0f, 0.0f, 6.0f, 3.0f};
__constant__ float     alphaTable3[4] = {4.0f, 0.0f, 2.0f, 2.0f};
__constant__ const int prods4[4]      = {0x090000, 0x000900, 0x040102, 0x010402};
__constant__ const int prods3[4]      = {0x040000, 0x000400, 0x040101, 0x010401};

#define USE_TABLES 1

static __device__ float
evalPermutation4(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    for (int i = 0; i < 16; i++) {
        const uint bits = permutation >> (2 * i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float  alpha2_sum    = float(akku >> 16);
    float  beta2_sum     = float((akku >> 8) & 0xff);
    float  alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum     = (9.0f * color_sum) - alphax_sum;
#else
    float  alpha2_sum    = 0.0f;
    float  beta2_sum     = 0.0f;
    float  alphabeta_sum = 0.0f;
    float3 alphax_sum    = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < 16; i++) {
        const uint bits = permutation >> (2 * i);

        float beta = (bits & 1);

        if (bits & 2) {
            beta = (1 + beta) * (1.0f / 3.0f);
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.111111111111f) * dot(e, kColorMetric);
}

static __device__ float
evalPermutation3(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    for (int i = 0; i < 16; i++) {
        const uint bits = permutation >> (2 * i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float  alpha2_sum    = float(akku >> 16);
    float  beta2_sum     = float((akku >> 8) & 0xff);
    float  alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum     = (4.0f * color_sum) - alphax_sum;
#else
    float  alpha2_sum    = 0.0f;
    float  beta2_sum     = 0.0f;
    float  alphabeta_sum = 0.0f;
    float3 alphax_sum    = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < 16; i++) {
        const uint bits = permutation >> (2 * i);

        float beta = (bits & 1);

        if (bits & 2) {
            beta = 0.5f;
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.25f) * dot(e, kColorMetric);
}

__device__ void evalAllPermutations(const float3    *colors,
                                    const uint      *permutations,
                                    ushort          &bestStart,
                                    ushort          &bestEnd,
                                    uint            &bestPermutation,
                                    float           *errors,
                                    float3           color_sum,
                                    cg::thread_block cta)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    __shared__ uint s_permutations[160];

    for (int i = 0; i < 16; i++) {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 992) {
            break;
        }

        ushort start, end;
        uint   permutation = permutations[pidx];

        if (pidx < 160) {
            s_permutations[pidx] = permutation;
        }

        float error = evalPermutation4(colors, permutation, &start, &end, color_sum);

        if (error < bestError) {
            bestError       = error;
            bestPermutation = permutation;
            bestStart       = start;
            bestEnd         = end;
        }
    }

    if (bestStart < bestEnd) {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;
    }

    cg::sync(cta);

    for (int i = 0; i < 3; i++) {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 160) {
            break;
        }

        ushort start, end;
        uint   permutation = s_permutations[pidx];
        float  error       = evalPermutation3(colors, permutation, &start, &end, color_sum);

        if (error < bestError) {
            bestError       = error;
            bestPermutation = permutation;
            bestStart       = start;
            bestEnd         = end;

            if (bestStart > bestEnd) {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;
            }
        }
    }

    errors[idx] = bestError;
}

__device__ int findMinError(float *errors, cg::thread_block cta)
{
    const int      idx = threadIdx.x;
    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

    cg::sync(cta);

    for (int d = NUM_THREADS / 2; d > 0; d >>= 1) {
        float err0   = errors[idx];
        float err1   = (idx + d) < NUM_THREADS ? errors[idx + d] : FLT_MAX;
        int   index1 = (idx + d) < NUM_THREADS ? indices[idx + d] : 0;

        cg::sync(cta);

        if (err1 < err0) {
            errors[idx]  = err1;
            indices[idx] = index1;
        }

        cg::sync(cta);
    }

    return indices[0];
}

__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 *result, int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;

    if (start == end) {
        permutation = 0;
    }

    uint indices = 0;

    for (int i = 0; i < 16; i++) {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }

    result[bid].x = (end << 16) | start;

    result[bid].y = indices;
}

__global__ void compress(const uint *permutations, const uint *image, uint2 *result, int blockOffset)
{
    cg::thread_block cta = cg::this_thread_block();

    const int idx = threadIdx.x;

    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int    xrefs[16];

    loadColorBlock(image, colors, sums, xrefs, blockOffset, cta);

    cg::sync(cta);

    ushort bestStart, bestEnd;
    uint   bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation, errors, sums[0], cta);

    const int minIdx = findMinError(errors, cta);

    cg::sync(cta);

    if (idx == minIdx) {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result, blockOffset);
    }
}

union Color32
{
    struct
    {
        unsigned char b, g, r, a;
    };
    unsigned int u;
};

union Color16
{
    struct
    {
        unsigned short b : 5;
        unsigned short g : 6;
        unsigned short r : 5;
    };
    unsigned short u;
};

struct BlockDXT1
{
    Color16 col0;
    Color16 col1;
    union
    {
        unsigned char row[4];
        unsigned int  indices;
    };

    void decompress(Color32 colors[16]) const;
};

void BlockDXT1::decompress(Color32 *colors) const
{
    Color32 palette[4];

    palette[0].b = (col0.b << 3) | (col0.b >> 2);
    palette[0].g = (col0.g << 2) | (col0.g >> 4);
    palette[0].r = (col0.r << 3) | (col0.r >> 2);
    palette[0].a = 0xFF;

    palette[1].r = (col1.r << 3) | (col1.r >> 2);
    palette[1].g = (col1.g << 2) | (col1.g >> 4);
    palette[1].b = (col1.b << 3) | (col1.b >> 2);
    palette[1].a = 0xFF;

    if (col0.u > col1.u) {
        palette[2].r = (2 * palette[0].r + palette[1].r) / 3;
        palette[2].g = (2 * palette[0].g + palette[1].g) / 3;
        palette[2].b = (2 * palette[0].b + palette[1].b) / 3;
        palette[2].a = 0xFF;

        palette[3].r = (2 * palette[1].r + palette[0].r) / 3;
        palette[3].g = (2 * palette[1].g + palette[0].g) / 3;
        palette[3].b = (2 * palette[1].b + palette[0].b) / 3;
        palette[3].a = 0xFF;
    }
    else {
        palette[2].r = (palette[0].r + palette[1].r) / 2;
        palette[2].g = (palette[0].g + palette[1].g) / 2;
        palette[2].b = (palette[0].b + palette[1].b) / 2;
        palette[2].a = 0xFF;

        palette[3].r = 0x00;
        palette[3].g = 0x00;
        palette[3].b = 0x00;
        palette[3].a = 0x00;
    }

    for (int i = 0; i < 16; i++) {
        colors[i] = palette[(indices >> (2 * i)) & 0x3];
    }
}

static int compareColors(const Color32 *b0, const Color32 *b1)
{
    int sum = 0;

    for (int i = 0; i < 16; i++) {
        int r = (b0[i].r - b1[i].r);
        int g = (b0[i].g - b1[i].g);
        int b = (b0[i].b - b1[i].b);
        sum += r * r + g * g + b * b;
    }

    return sum;
}

static int compareBlock(const BlockDXT1 *b0, const BlockDXT1 *b1)
{
    Color32 colors0[16];
    Color32 colors1[16];

    if (memcmp(b0, b1, sizeof(BlockDXT1)) == 0) {
        return 0;
    }
    else {
        b0->decompress(colors0);
        b1->decompress(colors1);

        return compareColors(colors0, colors1);
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    findCudaDevice(argc, (const char **)argv);

    unsigned char *data = NULL;
    uint           W, H;

    char *image_path = sdkFindFilePath(INPUT_IMAGE, argv[0]);

    if (image_path == 0) {
        printf("Error, unable to find source image  <%s>\n", image_path);
        exit(EXIT_FAILURE);
    }

    if (!sdkLoadPPM4ub(image_path, &data, &W, &H)) {
        printf("Error, unable to open source image file <%s>\n", image_path);

        exit(EXIT_FAILURE);
    }

    uint w = W, h = H;

    printf("Image Loaded '%s', %d x %d pixels\n\n", image_path, w, h);

    const uint memSize = w * h * 4;
    assert(0 != memSize);
    uint *block_image = (uint *)malloc(memSize);

    for (uint by = 0; by < h / 4; by++) {
        for (uint bx = 0; bx < w / 4; bx++) {
            for (int i = 0; i < 16; i++) {
                const int x                             = i & 3;
                const int y                             = i / 4;
                block_image[(by * w / 4 + bx) * 16 + i] = ((uint *)data)[(by * 4 + y) * 4 * (W / 4) + bx * 4 + x];
            }
        }
    }

    uint *d_data = NULL;
    checkCudaErrors(hipMalloc((void **)&d_data, memSize));

    uint      *d_result       = NULL;
    const uint compressedSize = (w / 4) * (h / 4) * 8;
    checkCudaErrors(hipMalloc((void **)&d_result, compressedSize));
    uint *h_result = (uint *)malloc(compressedSize);

    uint permutations[1024];
    computePermutations(permutations);

    uint *d_permutations = NULL;
    checkCudaErrors(hipMalloc((void **)&d_permutations, 1024 * sizeof(uint)));
    checkCudaErrors(hipMemcpy(d_permutations, permutations, 1024 * sizeof(uint), hipMemcpyHostToDevice));

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);

    checkCudaErrors(hipMemcpy(d_data, block_image, memSize, hipMemcpyHostToDevice));

    uint blocks = ((w + 3) / 4) * ((h + 3) / 4); 

    int            devID;
    hipDeviceProp_t deviceProp;

    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    int blocksPerLaunch = min(blocks, 768 * deviceProp.multiProcessorCount);

    printf("Running DXT Compression on %u x %u image...\n", w, h);
    printf("\n%u Blocks, %u Threads per Block, %u Threads in Grid...\n\n", blocks, NUM_THREADS, blocks * NUM_THREADS);
    int numIterations = 1;

    for (int i = -1; i < numIterations; ++i) {
        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkStartTimer(&timer);
        }

        for (int j = 0; j < (int)blocks; j += blocksPerLaunch) {
            compress<<<min(blocksPerLaunch, blocks - j), NUM_THREADS>>>(d_permutations, d_data, (uint2 *)d_result, j);
        }
    }

    getLastCudaError("compress");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    double dAvgTime = 1.0e-3 * sdkGetTimerValue(&timer) / (double)numIterations;
    printf("dxtc, Throughput = %.4f MPixels/s, Time = %.5f s, Size = %u Pixels, "
           "NumDevsUsed = %i, Workgroup = %d\n",
           (1.0e-6 * (double)(W * H) / dAvgTime),
           dAvgTime,
           (W * H),
           1,
           NUM_THREADS);

    checkCudaErrors(hipMemcpy(h_result, d_result, compressedSize, hipMemcpyDeviceToHost));

    char output_filename[1024];
    strcpy(output_filename, image_path);
    strcpy(output_filename + strlen(image_path) - 3, "dds");
    FILE *fp = fopen(output_filename, "wb");

    if (fp == 0) {
        printf("Error, unable to open output image <%s>\n", output_filename);
        exit(EXIT_FAILURE);
    }

    DDSHeader header;
    header.fourcc      = FOURCC_DDS;
    header.size        = 124;
    header.flags       = (DDSD_WIDTH | DDSD_HEIGHT | DDSD_CAPS | DDSD_PIXELFORMAT | DDSD_LINEARSIZE);
    header.height      = h;
    header.width       = w;
    header.pitch       = compressedSize;
    header.depth       = 0;
    header.mipmapcount = 0;
    memset(header.reserved, 0, sizeof(header.reserved));
    header.pf.size     = 32;
    header.pf.flags    = DDPF_FOURCC;
    header.pf.fourcc   = FOURCC_DXT1;
    header.pf.bitcount = 0;
    header.pf.rmask    = 0;
    header.pf.gmask    = 0;
    header.pf.bmask    = 0;
    header.pf.amask    = 0;
    header.caps.caps1  = DDSCAPS_TEXTURE;
    header.caps.caps2  = 0;
    header.caps.caps3  = 0;
    header.caps.caps4  = 0;
    header.notused     = 0;
    fwrite(&header, sizeof(DDSHeader), 1, fp);
    fwrite(h_result, compressedSize, 1, fp);
    fclose(fp);

    const char *reference_image_path = sdkFindFilePath(REFERENCE_IMAGE, argv[0]);

    if (reference_image_path == 0) {
        printf("Error, unable to find reference image\n");

        exit(EXIT_FAILURE);
    }

    fp = fopen(reference_image_path, "rb");

    if (fp == 0) {
        printf("Error, unable to open reference image\n");

        exit(EXIT_FAILURE);
    }

    fseek(fp, sizeof(DDSHeader), SEEK_SET);
    uint  referenceSize = (W / 4) * (H / 4) * 8;
    uint *reference     = (uint *)malloc(referenceSize);
    fread(reference, referenceSize, 1, fp);
    fclose(fp);

    printf("\nChecking accuracy...\n");
    float rms = 0;

    for (uint y = 0; y < h; y += 4) {
        for (uint x = 0; x < w; x += 4) {
            uint referenceBlockIdx = ((y / 4) * (W / 4) + (x / 4));
            uint resultBlockIdx    = ((y / 4) * (w / 4) + (x / 4));

            int cmp =
                compareBlock(((BlockDXT1 *)h_result) + resultBlockIdx, ((BlockDXT1 *)reference) + referenceBlockIdx);

            if (cmp != 0.0f) {
                printf("Deviation at (%4d,%4d):\t%f rms\n", x / 4, y / 4, float(cmp) / 16 / 3);
            }

            rms += cmp;
        }
    }

    rms /= w * h * 3;

    checkCudaErrors(hipFree(d_permutations));
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_result));
    free(image_path);
    free(data);
    free(block_image);
    free(h_result);
    free(reference);
    sdkDeleteTimer(&timer);

    printf("RMS(reference, result) = %f\n\n", rms);
    printf(rms <= ERROR_THRESHOLD ? "Test passed\n" : "Test failed!\n");
    return rms > ERROR_THRESHOLD;
}
