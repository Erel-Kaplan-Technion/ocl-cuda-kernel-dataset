#include "hip/hip_runtime.h"
#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;

hipTextureObject_t texObject;
hipTextureObject_t transferTex;

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;

struct Ray
{
    float3 o;
    float3 d;
};

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    float largest_tmin  = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar  = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

__device__ float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

__device__ float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) | (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void d_render(uint               *d_output,
                         uint                imageW,
                         uint                imageH,
                         float               density,
                         float               brightness,
                         float               transferOffset,
                         float               transferScale,
                         hipTextureObject_t tex,
                         hipTextureObject_t transferTex)
{
    const int    maxSteps         = 500;
    const float  tstep            = 0.01f;
    const float  opacityThreshold = 0.95f;
    const float3 boxMin           = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax           = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH))
        return;

    float u = (x / (float)imageW) * 2.0f - 1.0f;
    float v = (y / (float)imageH) * 2.0f - 1.0f;

    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    float tnear, tfar;
    int   hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit)
        return;

    if (tnear < 0.0f)
        tnear = 0.0f;

    float4 sum  = make_float4(0.0f);
    float  t    = tnear;
    float3 pos  = eyeRay.o + eyeRay.d * tnear;
    float3 step = eyeRay.d * tstep;

    for (int i = 0; i < maxSteps; i++) {
        float sample = tex3D<float>(tex, pos.x * 0.5f + 0.5f, pos.y * 0.5f + 0.5f, pos.z * 0.5f + 0.5f);

        float4 col = tex1D<float4>(transferTex, (sample - transferOffset) * transferScale);
        col.w *= density;

        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        sum = sum + col * (1.0f - sum.w);

        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar)
            break;

        pos += step;
    }

    sum *= brightness;

    d_output[y * imageW + x] = rgbaFloatToInt(sum);
}

extern "C" void setTextureFilterMode(bool bLinearFilter)
{
    if (texObject) {
        checkCudaErrors(hipDestroyTextureObject(texObject));
    }
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType         = hipResourceTypeArray;
    texRes.res.array.array = d_volumeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode       = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;

    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.addressMode[2] = hipAddressModeWrap;

    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));
}

extern "C" void initCuda(void *h_volume, hipExtent volumeSize)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr =
        make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType         = hipResourceTypeArray;
    texRes.res.array.array = d_volumeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode       = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;

    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));

    // create transfer function texture
    float4 transferFunc[] = {
        {
            0.0,
            0.0,
            0.0,
            0.0,
        },
        {
            1.0,
            0.0,
            0.0,
            1.0,
        },
        {
            1.0,
            0.5,
            0.0,
            1.0,
        },
        {
            1.0,
            1.0,
            0.0,
            1.0,
        },
        {
            0.0,
            1.0,
            0.0,
            1.0,
        },
        {
            0.0,
            1.0,
            1.0,
            1.0,
        },
        {
            0.0,
            0.0,
            1.0,
            1.0,
        },
        {
            1.0,
            0.0,
            1.0,
            1.0,
        },
        {
            0.0,
            0.0,
            0.0,
            0.0,
        },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray            *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1));
    checkCudaErrors(hipMemcpy2DToArray(
        d_transferFuncArray, 0, 0, transferFunc, 0, sizeof(transferFunc), 1, hipMemcpyHostToDevice));

    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType         = hipResourceTypeArray;
    texRes.res.array.array = d_transferFuncArray;

    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode       = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeClamp;

    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&transferTex, &texRes, &texDescr, NULL));
}

extern "C" void freeCudaBuffers()
{
    checkCudaErrors(hipDestroyTextureObject(texObject));
    checkCudaErrors(hipDestroyTextureObject(transferTex));
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}

extern "C" void render_kernel(dim3  gridSize,
                              dim3  blockSize,
                              uint *d_output,
                              uint  imageW,
                              uint  imageH,
                              float density,
                              float brightness,
                              float transferOffset,
                              float transferScale)
{
    d_render<<<gridSize, blockSize>>>(
        d_output, imageW, imageH, density, brightness, transferOffset, transferScale, texObject, transferTex);
}

extern "C" void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
