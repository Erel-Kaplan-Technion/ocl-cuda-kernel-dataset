#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>

#include "sortingNetworks_common.cuh"
#include "sortingNetworks_common.h"

__global__ void
bitonicSortShared(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint arrayLength, uint dir)
{
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x + 0]                       = d_SrcKey[0];
    s_val[threadIdx.x + 0]                       = d_SrcVal[0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < arrayLength; size <<= 1) {
        uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

        for (uint stride = size / 2; stride > 0; stride >>= 1) {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride], s_val[pos + stride], ddd);
        }
    }

    {
        for (uint stride = arrayLength / 2; stride > 0; stride >>= 1) {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride], s_val[pos + stride], dir);
        }
    }

    cg::sync(cta);
    d_DstKey[0]                       = s_key[threadIdx.x + 0];
    d_DstVal[0]                       = s_val[threadIdx.x + 0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

__global__ void bitonicSortShared1(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal)
{
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x + 0]                       = d_SrcKey[0];
    s_val[threadIdx.x + 0]                       = d_SrcVal[0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1) {
        uint ddd = (threadIdx.x & (size / 2)) != 0;

        for (uint stride = size / 2; stride > 0; stride >>= 1) {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride], s_val[pos + stride], ddd);
        }
    }

    uint ddd = blockIdx.x & 1;
    {
        for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1) {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride], s_val[pos + stride], ddd);
        }
    }

    cg::sync(cta);
    d_DstKey[0]                       = s_key[threadIdx.x + 0];
    d_DstVal[0]                       = s_val[threadIdx.x + 0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

__global__ void bitonicMergeGlobal(uint *d_DstKey,
                                   uint *d_DstVal,
                                   uint *d_SrcKey,
                                   uint *d_SrcVal,
                                   uint  arrayLength,
                                   uint  size,
                                   uint  stride,
                                   uint  dir)
{
    uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
    uint comparatorI        = global_comparatorI & (arrayLength / 2 - 1);

    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
    uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

    uint keyA = d_SrcKey[pos + 0];
    uint valA = d_SrcVal[pos + 0];
    uint keyB = d_SrcKey[pos + stride];
    uint valB = d_SrcVal[pos + stride];

    Comparator(keyA, valA, keyB, valB, ddd);

    d_DstKey[pos + 0]      = keyA;
    d_DstVal[pos + 0]      = valA;
    d_DstKey[pos + stride] = keyB;
    d_DstVal[pos + stride] = valB;
}

__global__ void bitonicMergeShared(uint *d_DstKey,
                                   uint *d_DstVal,
                                   uint *d_SrcKey,
                                   uint *d_SrcVal,
                                   uint  arrayLength,
                                   uint  size,
                                   uint  dir)
{
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x + 0]                       = d_SrcKey[0];
    s_val[threadIdx.x + 0]                       = d_SrcVal[0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    uint comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
    uint ddd         = dir ^ ((comparatorI & (size / 2)) != 0);

    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1) {
        cg::sync(cta);
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride], s_val[pos + stride], ddd);
    }

    cg::sync(cta);
    d_DstKey[0]                       = s_key[threadIdx.x + 0];
    d_DstVal[0]                       = s_val[threadIdx.x + 0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

extern "C" uint factorRadix2(uint *log2L, uint L)
{
    if (!L) {
        *log2L = 0;
        return 0;
    }
    else {
        for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++)
            ;

        return L;
    }
}

extern "C" uint
bitonicSort(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint batchSize, uint arrayLength, uint dir)
{
    if (arrayLength < 2)
        return 0;

    uint log2L;
    uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
    assert(factorizationRemainder == 1);

    dir = (dir != 0);

    uint blockCount  = batchSize * arrayLength / SHARED_SIZE_LIMIT;
    uint threadCount = SHARED_SIZE_LIMIT / 2;

    if (arrayLength <= SHARED_SIZE_LIMIT) {
        assert((batchSize * arrayLength) % SHARED_SIZE_LIMIT == 0);
        bitonicSortShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, dir);
    }
    else {
        bitonicSortShared1<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal);

        for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
            for (unsigned stride = size / 2; stride > 0; stride >>= 1)
                if (stride >= SHARED_SIZE_LIMIT) {
                    bitonicMergeGlobal<<<(batchSize * arrayLength) / 512, 256>>>(
                        d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride, dir);
                }
                else {
                    bitonicMergeShared<<<blockCount, threadCount>>>(
                        d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, dir);
                    break;
                }
    }

    return threadCount;
}
