#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#define PI 3.1415926535897932
#define BLOCK_X 16
#define BLOCK_Y 16

long M = INT_MAX;
int A = 1103515245;
int C = 12345;

const int threads_per_block = 128;

long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

float elapsed_time(long long start_time, long long end_time) {
        return (float) (end_time - start_time) / (1000 * 1000);
}

void check_error(hipError_t e) {
     if (e != hipSuccess) {
     	printf("\nCUDA error: %s\n", hipGetErrorString(e));
	    exit(1);
     }
}
__device__ int findIndexSeq(double * CDF, int lengthCDF, double value)
{
	int index = -1;
	int x;
	for(x = 0; x < lengthCDF; x++)
	{
		if(CDF[x] >= value)
		{
			index = x;
			break;
		}
	}
	if(index == -1)
		return lengthCDF-1;
	return index;
}
__device__ int findIndexBin(double * CDF, int beginIndex, int endIndex, double value)
{
	if(endIndex < beginIndex)
		return -1;
	int middleIndex;
	while(endIndex > beginIndex)
	{
		middleIndex = beginIndex + ((endIndex-beginIndex)/2);
		if(CDF[middleIndex] >= value)
		{
			if(middleIndex == 0)
				return middleIndex;
			else if(CDF[middleIndex-1] < value)
				return middleIndex;
			else if(CDF[middleIndex-1] == value)
			{
				while(CDF[middleIndex] == value && middleIndex >= 0)
					middleIndex--;
				middleIndex++;
				return middleIndex;
			}
		}
		if(CDF[middleIndex] > value)
			endIndex = middleIndex-1;
		else
			beginIndex = middleIndex+1;
	}
	return -1;
}

__global__ void kernel(double * arrayX, double * arrayY, double * CDF, double * u, double * xj, double * yj, int Nparticles){
	int block_id = blockIdx.x;// + gridDim.x * blockIdx.y;
	int i = blockDim.x * block_id + threadIdx.x;
	
	if(i < Nparticles){
	
		int index = -1;
		int x;
		
		for(x = 0; x < Nparticles; x++){
			if(CDF[x] >= u[i]){
				index = x;
				break;
			}
		}
		if(index == -1){
			index = Nparticles-1;
		}
		
		xj[i] = arrayX[index];
		yj[i] = arrayY[index];
		
	}
}

double roundDouble(double value){
	int newValue = (int)(value);
	if(value - newValue < .5)
	return newValue;
	else
	return newValue++;
}

void setIf(int testValue, int newValue, int * array3D, int * dimX, int * dimY, int * dimZ){
	int x, y, z;
	for(x = 0; x < *dimX; x++){
		for(y = 0; y < *dimY; y++){
			for(z = 0; z < *dimZ; z++){
				if(array3D[x * *dimY * *dimZ+y * *dimZ + z] == testValue)
				array3D[x * *dimY * *dimZ + y * *dimZ + z] = newValue;
			}
		}
	}
}

double randu(int * seed, int index)
{
	int num = A*seed[index] + C;
	seed[index] = num % M;
	return fabs(seed[index]/((double) M));
}

double randn(int * seed, int index){
	double u = randu(seed, index);
	double v = randu(seed, index);
	double cosine = cos(2*PI*v);
	double rt = -2*log(u);
	return sqrt(rt)*cosine;
}

void addNoise(int * array3D, int * dimX, int * dimY, int * dimZ, int * seed){
	int x, y, z;
	for(x = 0; x < *dimX; x++){
		for(y = 0; y < *dimY; y++){
			for(z = 0; z < *dimZ; z++){
				array3D[x * *dimY * *dimZ + y * *dimZ + z] = array3D[x * *dimY * *dimZ + y * *dimZ + z] + (int)(5*randn(seed, 0));
			}
		}
	}
}

void strelDisk(int * disk, int radius)
{
	int diameter = radius*2 - 1;
	int x, y;
	for(x = 0; x < diameter; x++){
		for(y = 0; y < diameter; y++){
			double distance = sqrt(pow((double)(x-radius+1),2) + pow((double)(y-radius+1),2));
			if(distance < radius)
			    disk[x*diameter + y] = 1;
            else
			    disk[x*diameter + y] = 0;
		}
	}
}

void dilate_matrix(int * matrix, int posX, int posY, int posZ, int dimX, int dimY, int dimZ, int error)
{
	int startX = posX - error;
	while(startX < 0)
	startX++;
	int startY = posY - error;
	while(startY < 0)
	startY++;
	int endX = posX + error;
	while(endX > dimX)
	endX--;
	int endY = posY + error;
	while(endY > dimY)
	endY--;
	int x,y;
	for(x = startX; x < endX; x++){
		for(y = startY; y < endY; y++){
			double distance = sqrt( pow((double)(x-posX),2) + pow((double)(y-posY),2) );
			if(distance < error)
			matrix[x*dimY*dimZ + y*dimZ + posZ] = 1;
		}
	}
}


void imdilate_disk(int * matrix, int dimX, int dimY, int dimZ, int error, int * newMatrix)
{
	int x, y, z;
	for(z = 0; z < dimZ; z++){
		for(x = 0; x < dimX; x++){
			for(y = 0; y < dimY; y++){
				if(matrix[x*dimY*dimZ + y*dimZ + z] == 1){
					dilate_matrix(newMatrix, x, y, z, dimX, dimY, dimZ, error);
				}
			}
		}
	}
}

void getneighbors(int * se, int numOnes, double * neighbors, int radius){
	int x, y;
	int neighY = 0;
	int center = radius - 1;
	int diameter = radius*2 -1;
	for(x = 0; x < diameter; x++){
		for(y = 0; y < diameter; y++){
			if(se[x*diameter + y]){
				neighbors[neighY*2] = (int)(y - center);
				neighbors[neighY*2 + 1] = (int)(x - center);
				neighY++;
			}
		}
	}
}

void videoSequence(int * I, int IszX, int IszY, int Nfr, int * seed){
	int k;
	int max_size = IszX*IszY*Nfr;
	int x0 = (int)roundDouble(IszY/2.0);
	int y0 = (int)roundDouble(IszX/2.0);
	I[x0 *IszY *Nfr + y0 * Nfr  + 0] = 1;
	
	int xk, yk, pos;
	for(k = 1; k < Nfr; k++){
		xk = abs(x0 + (k-1));
		yk = abs(y0 - 2*(k-1));
		pos = yk * IszY * Nfr + xk *Nfr + k;
		if(pos >= max_size)
		pos = 0;
		I[pos] = 1;
	}
	
	int * newMatrix = (int *)malloc(sizeof(int)*IszX*IszY*Nfr);
	imdilate_disk(I, IszX, IszY, Nfr, 5, newMatrix);
	int x, y;
	for(x = 0; x < IszX; x++){
		for(y = 0; y < IszY; y++){
			for(k = 0; k < Nfr; k++){
				I[x*IszY*Nfr + y*Nfr + k] = newMatrix[x*IszY*Nfr + y*Nfr + k];
			}
		}
	}
	free(newMatrix);
	
	setIf(0, 100, I, &IszX, &IszY, &Nfr);
	setIf(1, 228, I, &IszX, &IszY, &Nfr);
	addNoise(I, &IszX, &IszY, &Nfr, seed);
}

double calcLikelihoodSum(int * I, int * ind, int numOnes){
	double likelihoodSum = 0.0;
	int y;
	for(y = 0; y < numOnes; y++)
	likelihoodSum += (pow((double)(I[ind[y]] - 100),2) - pow((double)(I[ind[y]]-228),2))/50.0;
	return likelihoodSum;
}

int findIndex(double * CDF, int lengthCDF, double value){
	int index = -1;
	int x;
	for(x = 0; x < lengthCDF; x++){
		if(CDF[x] >= value){
			index = x;
			break;
		}
	}
	if(index == -1){
		return lengthCDF-1;
	}
	return index;
}

void particleFilter(int * I, int IszX, int IszY, int Nfr, int * seed, int Nparticles){
	int max_size = IszX*IszY*Nfr;
	long long start = get_time();
	double xe = roundDouble(IszY/2.0);
	double ye = roundDouble(IszX/2.0);
	
	int radius = 5;
	int diameter = radius*2 - 1;
	int * disk = (int *)malloc(diameter*diameter*sizeof(int));
	strelDisk(disk, radius);
	int countOnes = 0;
	int x, y;
	for(x = 0; x < diameter; x++){
		for(y = 0; y < diameter; y++){
			if(disk[x*diameter + y] == 1)
				countOnes++;
		}
	}
	double * objxy = (double *)malloc(countOnes*2*sizeof(double));
	getneighbors(disk, countOnes, objxy, radius);
	
	long long get_neighbors = get_time();
	printf("TIME TO GET NEIGHBORS TOOK: %f\n", elapsed_time(start, get_neighbors));
	double * weights = (double *)malloc(sizeof(double)*Nparticles);
	for(x = 0; x < Nparticles; x++){
		weights[x] = 1/((double)(Nparticles));
	}
	long long get_weights = get_time();
	printf("TIME TO GET WEIGHTSTOOK: %f\n", elapsed_time(get_neighbors, get_weights));
	double * likelihood = (double *)malloc(sizeof(double)*Nparticles);
	double * arrayX = (double *)malloc(sizeof(double)*Nparticles);
	double * arrayY = (double *)malloc(sizeof(double)*Nparticles);
	double * xj = (double *)malloc(sizeof(double)*Nparticles);
	double * yj = (double *)malloc(sizeof(double)*Nparticles);
	double * CDF = (double *)malloc(sizeof(double)*Nparticles);
	
	double * arrayX_GPU;
	double * arrayY_GPU;
	double * xj_GPU;
	double * yj_GPU;
	double * CDF_GPU;
	
	int * ind = (int*)malloc(sizeof(int)*countOnes);
	double * u = (double *)malloc(sizeof(double)*Nparticles);
	double * u_GPU;
	
	check_error(hipMalloc((void **) &arrayX_GPU, sizeof(double)*Nparticles));
	check_error(hipMalloc((void **) &arrayY_GPU, sizeof(double)*Nparticles));
	check_error(hipMalloc((void **) &xj_GPU, sizeof(double)*Nparticles));
	check_error(hipMalloc((void **) &yj_GPU, sizeof(double)*Nparticles));
	check_error(hipMalloc((void **) &CDF_GPU, sizeof(double)*Nparticles));
	check_error(hipMalloc((void **) &u_GPU, sizeof(double)*Nparticles));
	
	for(x = 0; x < Nparticles; x++){
		arrayX[x] = xe;
		arrayY[x] = ye;
	}
	int k;
	int indX, indY;
	for(k = 1; k < Nfr; k++){
		long long set_arrays = get_time();
		
		for(x = 0; x < Nparticles; x++){
			arrayX[x] = arrayX[x] + 1.0 + 5.0*randn(seed, x);
			arrayY[x] = arrayY[x] - 2.0 + 2.0*randn(seed, x);
		}
		long long error = get_time();
		printf("TIME TO SET ERROR TOOK: %f\n", elapsed_time(set_arrays, error));
		for(x = 0; x < Nparticles; x++){
		
			for(y = 0; y < countOnes; y++){
				indX = roundDouble(arrayX[x]) + objxy[y*2 + 1];
				indY = roundDouble(arrayY[x]) + objxy[y*2];
				ind[y] = fabs(indX*IszY*Nfr + indY*Nfr + k);
				if(ind[y] >= max_size)
					ind[y] = 0;
			}
			likelihood[x] = calcLikelihoodSum(I, ind, countOnes);
			likelihood[x] = likelihood[x]/countOnes;
		}
		long long likelihood_time = get_time();
		printf("TIME TO GET LIKELIHOODS TOOK: %f\n", elapsed_time(error, likelihood_time));
		for(x = 0; x < Nparticles; x++){
			weights[x] = weights[x] * exp(likelihood[x]);
		}
		long long exponential = get_time();
		printf("TIME TO GET EXP TOOK: %f\n", elapsed_time(likelihood_time, exponential));
		double sumWeights = 0;	
		for(x = 0; x < Nparticles; x++){
			sumWeights += weights[x];
		}
		long long sum_time = get_time();
		printf("TIME TO SUM WEIGHTS TOOK: %f\n", elapsed_time(exponential, sum_time));
		for(x = 0; x < Nparticles; x++){
				weights[x] = weights[x]/sumWeights;
		}
		long long normalize = get_time();
		printf("TIME TO NORMALIZE WEIGHTS TOOK: %f\n", elapsed_time(sum_time, normalize));
		xe = 0;
		ye = 0;
		for(x = 0; x < Nparticles; x++){
			xe += arrayX[x] * weights[x];
			ye += arrayY[x] * weights[x];
		}
		long long move_time = get_time();
		printf("TIME TO MOVE OBJECT TOOK: %f\n", elapsed_time(normalize, move_time));
		printf("XE: %lf\n", xe);
		printf("YE: %lf\n", ye);
		double distance = sqrt( pow((double)(xe-(int)roundDouble(IszY/2.0)),2) + pow((double)(ye-(int)roundDouble(IszX/2.0)),2) );
		printf("%lf\n", distance);
		
		
		CDF[0] = weights[0];
		for(x = 1; x < Nparticles; x++){
			CDF[x] = weights[x] + CDF[x-1];
		}
		long long cum_sum = get_time();
		printf("TIME TO CALC CUM SUM TOOK: %f\n", elapsed_time(move_time, cum_sum));
		double u1 = (1/((double)(Nparticles)))*randu(seed, 0);
		for(x = 0; x < Nparticles; x++){
			u[x] = u1 + x/((double)(Nparticles));
		}
		long long u_time = get_time();
		printf("TIME TO CALC U TOOK: %f\n", elapsed_time(cum_sum, u_time));
		long long start_copy = get_time();
		hipMemcpy(arrayX_GPU, arrayX, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		hipMemcpy(arrayY_GPU, arrayY, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		hipMemcpy(xj_GPU, xj, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		hipMemcpy(yj_GPU, yj, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		hipMemcpy(CDF_GPU, CDF, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		hipMemcpy(u_GPU, u, sizeof(double)*Nparticles, hipMemcpyHostToDevice);
		long long end_copy = get_time();
		int num_blocks = ceil((double) Nparticles/(double) threads_per_block);
		
		kernel <<< num_blocks, threads_per_block >>> (arrayX_GPU, arrayY_GPU, CDF_GPU, u_GPU, xj_GPU, yj_GPU, Nparticles);
                hipDeviceSynchronize();
                long long start_copy_back = get_time();
		hipMemcpy(yj, yj_GPU, sizeof(double)*Nparticles, hipMemcpyDeviceToHost);
		hipMemcpy(xj, xj_GPU, sizeof(double)*Nparticles, hipMemcpyDeviceToHost);
		long long end_copy_back = get_time();
		printf("SENDING TO GPU TOOK: %lf\n", elapsed_time(start_copy, end_copy));
		printf("CUDA EXEC TOOK: %lf\n", elapsed_time(end_copy, start_copy_back));
		printf("SENDING BACK FROM GPU TOOK: %lf\n", elapsed_time(start_copy_back, end_copy_back));
		long long xyj_time = get_time();
		printf("TIME TO CALC NEW ARRAY X AND Y TOOK: %f\n", elapsed_time(u_time, xyj_time));
		
		for(x = 0; x < Nparticles; x++){
			arrayX[x] = xj[x];
			arrayY[x] = yj[x];
			weights[x] = 1/((double)(Nparticles));
		}
		long long reset = get_time();
		printf("TIME TO RESET WEIGHTS TOOK: %f\n", elapsed_time(xyj_time, reset));
	}
	
	hipFree(u_GPU);
	hipFree(CDF_GPU);
	hipFree(yj_GPU);
	hipFree(xj_GPU);
	hipFree(arrayY_GPU);
	hipFree(arrayX_GPU);
	
	free(disk);
	free(objxy);
	free(weights);
	free(likelihood);
	free(arrayX);
	free(arrayY);
	free(xj);
	free(yj);
	free(CDF);
	free(u);
	free(ind);
}
int main(int argc, char * argv[]){
	
	char* usage = "naive.out -x <dimX> -y <dimY> -z <Nfr> -np <Nparticles>";
	if(argc != 9)
	{
		printf("%s\n", usage);
		return 0;
	}
	if( strcmp( argv[1], "-x" ) ||  strcmp( argv[3], "-y" ) || strcmp( argv[5], "-z" ) || strcmp( argv[7], "-np" ) ) {
		printf( "%s\n",usage );
		return 0;
	}
	
	int IszX, IszY, Nfr, Nparticles;
	
	if( sscanf( argv[2], "%d", &IszX ) == EOF ) {
	   printf("ERROR: dimX input is incorrect");
	   return 0;
	}
	
	if( IszX <= 0 ) {
		printf("dimX must be > 0\n");
		return 0;
	}
	
	if( sscanf( argv[4], "%d", &IszY ) == EOF ) {
	   printf("ERROR: dimY input is incorrect");
	   return 0;
	}
	
	if( IszY <= 0 ) {
		printf("dimY must be > 0\n");
		return 0;
	}
	
	if( sscanf( argv[6], "%d", &Nfr ) == EOF ) {
	   printf("ERROR: Number of frames input is incorrect");
	   return 0;
	}
	
	if( Nfr <= 0 ) {
		printf("number of frames must be > 0\n");
		return 0;
	}
	
	if( sscanf( argv[8], "%d", &Nparticles ) == EOF ) {
	   printf("ERROR: Number of particles input is incorrect");
	   return 0;
	}
	
	if( Nparticles <= 0 ) {
		printf("Number of particles must be > 0\n");
		return 0;
	}
	int * seed = (int *)malloc(sizeof(int)*Nparticles);
	int i;
	for(i = 0; i < Nparticles; i++)
		seed[i] = time(0)*i;
	int * I = (int *)malloc(sizeof(int)*IszX*IszY*Nfr);
	long long start = get_time();
	videoSequence(I, IszX, IszY, Nfr, seed);
	long long endVideoSequence = get_time();
	printf("VIDEO SEQUENCE TOOK %f\n", elapsed_time(start, endVideoSequence));
	particleFilter(I, IszX, IszY, Nfr, seed, Nparticles);
	long long endParticleFilter = get_time();
	printf("PARTICLE FILTER TOOK %f\n", elapsed_time(endVideoSequence, endParticleFilter));
	printf("ENTIRE PROGRAM TOOK %f\n", elapsed_time(start, endParticleFilter));
	
	free(seed);
	free(I);
	return 0;
}
