#include "hip/hip_runtime.h"
#include "streamcluster_header.cu"

using namespace std;

#define CUDA_SAFE_CALL( call) do {										\
   hipError_t err = call;												\
   if( hipSuccess != err) {											\
       fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
               __FILE__, __LINE__, hipGetErrorString( err) );			\
   exit(EXIT_FAILURE);													\
   } } while (0)

#define THREADS_PER_BLOCK 512
#define MAXBLOCKS 65536
#define CUDATIME

float *work_mem_h;
float *coord_h;

float *work_mem_d;
float *coord_d;
int   *center_table_d;
bool  *switch_membership_d;
Point *p;

static int iter = 0;		// counter for total# of iteration


__device__ float
d_dist(int p1, int p2, int num, int dim, float *coord_d)
{
	float retval = 0.0;
	for(int i = 0; i < dim; i++){
		float tmp = coord_d[(i*num)+p1] - coord_d[(i*num)+p2];
		retval += tmp * tmp;
	}
	return retval;
}

__global__ void
kernel_compute_cost(int num, int dim, long x, Point *p, int K, int stride,
					float *coord_d, float *work_mem_d, int *center_table_d, bool *switch_membership_d)
{
	// block ID and global thread ID
	const int bid  = blockIdx.x + gridDim.x * blockIdx.y;
	const int tid = blockDim.x * bid + threadIdx.x;

	if(tid < num)
	{
		float *lower = &work_mem_d[tid*stride];
		
		// cost between this point and point[x]: euclidean distance multiplied by weight
		float x_cost = d_dist(tid, x, num, dim, coord_d) * p[tid].weight;
		
		// if computed cost is less then original (it saves), mark it as to reassign
		if ( x_cost < p[tid].cost )
		{
			switch_membership_d[tid] = 1;
			lower[K] += x_cost - p[tid].cost;
		}
		// if computed cost is larger, save the difference
		else
		{
			lower[center_table_d[p[tid].assign]] += p[tid].cost - x_cost;
		}
	}
}

void allocDevMem(int num, int dim)
{
	CUDA_SAFE_CALL( hipMalloc((void**) &center_table_d,	  num * sizeof(int))   );
	CUDA_SAFE_CALL( hipMalloc((void**) &switch_membership_d, num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMalloc((void**) &p,					  num * sizeof(Point)) );
	CUDA_SAFE_CALL( hipMalloc((void**) &coord_d,		num * dim * sizeof(float)) );
}

void allocHostMem(int num, int dim)
{
	coord_h	= (float*) malloc( num * dim * sizeof(float) );
}

void freeDevMem()
{
	CUDA_SAFE_CALL( hipFree(center_table_d)	  );
	CUDA_SAFE_CALL( hipFree(switch_membership_d) );
	CUDA_SAFE_CALL( hipFree(p)					  );
	CUDA_SAFE_CALL( hipFree(coord_d)			  );
}

void freeHostMem()
{
	free(coord_h);
}

float pgain( long x, Points *points, float z, long int *numcenters, int kmax, bool *is_center, int *center_table, bool *switch_membership, bool isCoordChanged,
							double *serial_t, double *cpu_to_gpu_t, double *gpu_to_cpu_t, double *alloc_t, double *kernel_t, double *free_t)
{	
#ifdef CUDATIME
	float tmp_t;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
#endif

	hipError_t error;
	
	int stride	= *numcenters + 1;			// size of each work_mem segment
	int K		= *numcenters ;				// number of centers
	int num		=  points->num;				// number of points
	int dim		=  points->dim;				// number of dimension
	int nThread =  num;						// number of threads == number of data points

	work_mem_h = (float*) malloc(stride * (nThread + 1) * sizeof(float) );
	if(iter == 0)
	{
		allocHostMem(num, dim);
	}
	
	int count = 0;
	for( int i=0; i<num; i++)
	{
		if( is_center[i] )
		{
			center_table[i] = count++;
		}
	}

	if(isCoordChanged || iter == 0)
	{
		for(int i=0; i<dim; i++)
		{
			for(int j=0; j<num; j++)
			{
				coord_h[ (num*i)+j ] = points->p[j].coord[i];
			}
		}
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	CUDA_SAFE_CALL( hipMalloc((void**) &work_mem_d,  stride * (nThread + 1) * sizeof(float)) );
	if( iter == 0 )
	{
		allocDevMem(num, dim);
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*alloc_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	if(isCoordChanged || iter == 0)
	{
		CUDA_SAFE_CALL( hipMemcpy(coord_d,  coord_h,	 num * dim * sizeof(float), hipMemcpyHostToDevice) );
	}
	CUDA_SAFE_CALL( hipMemcpy(center_table_d,  center_table,  num * sizeof(int),   hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(p,  points->p,				   num * sizeof(Point), hipMemcpyHostToDevice) );
	
	CUDA_SAFE_CALL( hipMemset((void*) switch_membership_d, 0,			num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMemset((void*) work_mem_d,  		0, stride * (nThread + 1) * sizeof(float)) );
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*cpu_to_gpu_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	// Determine the number of thread blocks in the x- and y-dimension
	int num_blocks 	 = (int) ((float) (num + THREADS_PER_BLOCK - 1) / (float) THREADS_PER_BLOCK);
	int num_blocks_y = (int) ((float) (num_blocks + MAXBLOCKS - 1)  / (float) MAXBLOCKS);
	int num_blocks_x = (int) ((float) (num_blocks+num_blocks_y - 1) / (float) num_blocks_y);	
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);

	kernel_compute_cost<<<grid_size, THREADS_PER_BLOCK>>>(	
															num,					// in:	# of data
															dim,					// in:	dimension of point coordinates
															x,						// in:	point to open a center at
															p,						// in:	data point array
															K,						// in:	number of centers
															stride,					// in:  size of each work_mem segment
															coord_d,				// in:	array of point coordinates
															work_mem_d,				// out:	cost and lower field array
															center_table_d,			// in:	center index table
															switch_membership_d		// out:  changes in membership
															);
	hipDeviceSynchronize();
	
	// error check
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*kernel_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	CUDA_SAFE_CALL( hipMemcpy(work_mem_h, 		  work_mem_d, 	stride * (nThread + 1) * sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(switch_membership, switch_membership_d,	 num * sizeof(bool),  hipMemcpyDeviceToHost) );
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*gpu_to_cpu_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	int number_of_centers_to_close = 0;
	float gl_cost_of_opening_x = z;
	float *gl_lower = &work_mem_h[stride * nThread];
	for(int i=0; i < num; i++)
	{
		if( is_center[i] )
		{
			float low = z;
		    for( int j = 0; j < num; j++ )
			{
				low += work_mem_h[ j*stride + center_table[i] ];
			}
			
		    gl_lower[center_table[i]] = low;
				
		    if ( low > 0 )
			{
				++number_of_centers_to_close;
				work_mem_h[i*stride+K] -= low;
		    }
		}
		gl_cost_of_opening_x += work_mem_h[i*stride+K];
	}

	if ( gl_cost_of_opening_x < 0 )
	{
		for(int i = 0; i < num; i++)
		{
			bool close_center = gl_lower[center_table[points->p[i].assign]] > 0 ;
			if ( switch_membership[i] || close_center )
			{
				points->p[i].cost = dist(points->p[i], points->p[x], dim) * points->p[i].weight;
				points->p[i].assign = x;
			}
		}
		
		for(int i = 0; i < num; i++)
		{
			if( is_center[i] && gl_lower[center_table[i]] > 0 )
			{
				is_center[i] = false;
			}
		}
		
		if( x >= 0 && x < num)
		{
			is_center[x] = true;
		}
		*numcenters = *numcenters + 1 - number_of_centers_to_close;
	}
	else
	{
		gl_cost_of_opening_x = 0;
	}

	free(work_mem_h);

	// free device memory
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	CUDA_SAFE_CALL( hipFree(work_mem_d) );
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*free_t += (double) tmp_t;
#endif
	iter++;
	return -gl_cost_of_opening_x;
}
