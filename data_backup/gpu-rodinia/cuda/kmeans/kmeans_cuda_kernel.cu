#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

#define ASSUMED_NR_CLUSTERS 32

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

texture<float, 1, hipReadModeElementType> t_features;
texture<float, 1, hipReadModeElementType> t_features_flipped;
texture<float, 1, hipReadModeElementType> t_clusters;


__constant__ float c_clusters[ASSUMED_NR_CLUSTERS*34];


__global__ void invert_mapping(float *input,
							   float *output,
							   int npoints,
							   int nfeatures)
{
	int point_id = threadIdx.x + blockDim.x*blockIdx.x;
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++)
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
	}
	return;
}


__global__ void
kmeansPoint(float  *features,
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			float  *clusters,
			float  *block_clusters,
			int    *block_deltas) 
{

	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;
  
	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		float min_dist = FLT_MAX;
		float dist;
		
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;
			float ans=0.0;

			for (j=0; j < nfeatures; j++)
			{					
				int addr = point_id + j*npoints;
				float diff = (tex1Dfetch(t_features,addr) -
							  c_clusters[cluster_base_index + j]);
				ans += diff*diff;
			}
			dist = ans;		

			if (dist < min_dist) {
				min_dist = dist;
				index    = i;
			}
		}
	}
	

#ifdef GPU_DELTA_REDUCTION
	__shared__ int deltas[THREADS_PER_BLOCK];
	if(threadIdx.x < THREADS_PER_BLOCK) {
		deltas[threadIdx.x] = 0;
	}
#endif
	if (point_id < npoints)
	{
#ifdef GPU_DELTA_REDUCTION
		if (membership[point_id] != index) {
			deltas[threadIdx.x] = 1;
		}
#endif
		membership[point_id] = index;
	}

#ifdef GPU_DELTA_REDUCTION
	__syncthreads();

	unsigned int threadids_participating = THREADS_PER_BLOCK / 2;
	for(;threadids_participating > 1; threadids_participating /= 2) {
   		if(threadIdx.x < threadids_participating) {
			deltas[threadIdx.x] += deltas[threadIdx.x + threadids_participating];
		}
   		__syncthreads();
	}
	if(threadIdx.x < 1)	{deltas[threadIdx.x] += deltas[threadIdx.x + 1];}
	__syncthreads();
	if(threadIdx.x == 0) {
		block_deltas[blockIdx.y * gridDim.x + blockIdx.x] = deltas[0];
		
	}

#endif


#ifdef GPU_NEW_CENTER_REDUCTION
	int center_id = threadIdx.x / nfeatures;    
	int dim_id = threadIdx.x - nfeatures*center_id;

	__shared__ int new_center_ids[THREADS_PER_BLOCK];

	new_center_ids[threadIdx.x] = index;
	__syncthreads();

	int new_base_index = (point_id - threadIdx.x)*nfeatures + dim_id;
	float accumulator = 0.f;

	if(threadIdx.x < nfeatures * nclusters) {
		for(int i = 0; i< (THREADS_PER_BLOCK); i++) {
			float val = tex1Dfetch(t_features_flipped,new_base_index+i*nfeatures);
			if(new_center_ids[i] == center_id) 
				accumulator += val;
		}
	
		block_clusters[(blockIdx.y*gridDim.x + blockIdx.x) * nclusters * nfeatures + threadIdx.x] = accumulator;
	}
#endif

}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_
