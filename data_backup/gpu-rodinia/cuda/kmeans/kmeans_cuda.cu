#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

#include <omp.h>

#include <hip/hip_runtime.h>

#define THREADS_PER_DIM 16
#define BLOCKS_PER_DIM 16
#define THREADS_PER_BLOCK THREADS_PER_DIM*THREADS_PER_DIM

#include "kmeans_cuda_kernel.cu"


#define CPU_DELTA_REDUCE
#define CPU_CENTER_REDUCE

extern "C"
int setup(int argc, char** argv);									/* function prototype */

unsigned int num_threads_perdim = THREADS_PER_DIM;					/* sqrt(256) -- see references for this choice */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;					/* temporary */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	/* number of threads */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		/* number of blocks */

int    *membership_new;												/* newly assignment membership */
float  *feature_d;													/* inverted data array */
float  *feature_flipped_d;											/* original (not inverted) data array */
int    *membership_d;												/* membership on the device */
float  *block_new_centers;											/* sum of points in a cluster (per block) */
float  *clusters_d;													/* cluster centers on the device */
float  *block_clusters_d;											/* per block calculation of cluster centers */
int    *block_deltas_d;												/* per block calculation of deltas */


extern "C"
void allocateMemory(int npoints, int nfeatures, int nclusters, float **features)
{	
	num_blocks = npoints / num_threads;
	if (npoints % num_threads > 0)
		num_blocks++;

	num_blocks_perdim = sqrt((double) num_blocks);
	while (num_blocks_perdim * num_blocks_perdim < num_blocks)
		num_blocks_perdim++;

	num_blocks = num_blocks_perdim*num_blocks_perdim;

	membership_new = (int*) malloc(npoints * sizeof(int));
	for(int i=0;i<npoints;i++) {
		membership_new[i] = -1;
	}

	block_new_centers = (float *) malloc(nclusters*nfeatures*sizeof(float));
	
	hipMalloc((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float));
	hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**) &feature_d, npoints*nfeatures*sizeof(float));
		
	invert_mapping<<<num_blocks,num_threads>>>(feature_flipped_d,feature_d,npoints,nfeatures);
		
	hipMalloc((void**) &membership_d, npoints*sizeof(int));
	hipMalloc((void**) &clusters_d, nclusters*nfeatures*sizeof(float));

	
#ifdef BLOCK_DELTA_REDUCE
	hipMalloc((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int));
#endif

#ifdef BLOCK_CENTER_REDUCE
	hipMalloc((void**) &block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
#endif

}

extern "C"
void deallocateMemory()
{
	free(membership_new);
	free(block_new_centers);
	hipFree(feature_d);
	hipFree(feature_flipped_d);
	hipFree(membership_d);

	hipFree(clusters_d);
#ifdef BLOCK_CENTER_REDUCE
    hipFree(block_clusters_d);
#endif
#ifdef BLOCK_DELTA_REDUCE
    hipFree(block_deltas_d);
#endif
}

int
main( int argc, char** argv) 
{
    hipSetDevice(1);
	setup(argc, argv);    
}

extern "C"
int
kmeansCuda(float  **feature,
           int      nfeatures,
           int      npoints,
           int      nclusters,
           int     *membership,
		   float  **clusters,
		   int     *new_centers_len,
           float  **new_centers
		   )
{
	int delta = 0;			/* if point has moved */
	int i,j;				/* counters */


	hipSetDevice(1);

	hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);

    hipChannelFormatDesc chDesc0 = hipCreateChannelDesc<float>();
    t_features.filterMode = hipFilterModePoint;   
    t_features.normalized = false;
    t_features.channelDesc = chDesc0;

	if(hipBindTexture(NULL, &t_features, feature_d, &chDesc0, npoints*nfeatures*sizeof(float)) != hipSuccess)
        printf("Couldn't bind features array to texture!\n");

	hipChannelFormatDesc chDesc1 = hipCreateChannelDesc<float>();
    t_features_flipped.filterMode = hipFilterModePoint;   
    t_features_flipped.normalized = false;
    t_features_flipped.channelDesc = chDesc1;

	if(hipBindTexture(NULL, &t_features_flipped, feature_flipped_d, &chDesc1, npoints*nfeatures*sizeof(float)) != hipSuccess)
        printf("Couldn't bind features_flipped array to texture!\n");

	hipChannelFormatDesc chDesc2 = hipCreateChannelDesc<float>();
    t_clusters.filterMode = hipFilterModePoint;   
    t_clusters.normalized = false;
    t_clusters.channelDesc = chDesc2;

	if(hipBindTexture(NULL, &t_clusters, clusters_d, &chDesc2, nclusters*nfeatures*sizeof(float)) != hipSuccess)
        printf("Couldn't bind clusters array to texture!\n");

	hipMemcpyToSymbol(HIP_SYMBOL("c_clusters"),clusters[0],nclusters*nfeatures*sizeof(float),0,hipMemcpyHostToDevice);


    dim3  grid( num_blocks_perdim, num_blocks_perdim );
    dim3  threads( num_threads_perdim*num_threads_perdim );
    
    kmeansPoint<<< grid, threads >>>( feature_d,
                                      nfeatures,
                                      npoints,
                                      nclusters,
                                      membership_d,
                                      clusters_d,
									  block_clusters_d,
									  block_deltas_d);

	hipDeviceSynchronize();

	hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost);	

#ifdef BLOCK_CENTER_REDUCE
    float * block_clusters_h = (float *) malloc(
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
        
	hipMemcpy(block_clusters_h, block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float), 
        hipMemcpyDeviceToHost);
#endif
#ifdef BLOCK_DELTA_REDUCE
    int * block_deltas_h = (int *) malloc(
        num_blocks_perdim * num_blocks_perdim * sizeof(int));
        
	hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost);
#endif
    
	delta = 0;
	for (i = 0; i < npoints; i++)
	{		
		int cluster_id = membership_new[i];
		new_centers_len[cluster_id]++;
		if (membership_new[i] != membership[i])
		{
#ifdef CPU_DELTA_REDUCE
			delta++;
#endif
			membership[i] = membership_new[i];
		}
#ifdef CPU_CENTER_REDUCE
		for (j = 0; j < nfeatures; j++)
		{			
			new_centers[cluster_id][j] += feature[i][j];
		}
#endif
	}
	

#ifdef BLOCK_DELTA_REDUCE	
    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
        delta += block_deltas_h[i];
    }
        
#endif
#ifdef BLOCK_CENTER_REDUCE	
	
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++) {
			block_new_centers[j*nfeatures + k] = 0.f;
		}
	}

    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
		for(int j = 0; j < nclusters;j++) {
			for(int k = 0; k < nfeatures;k++) {
				block_new_centers[j*nfeatures + k] += block_clusters_h[i * nclusters*nfeatures + j * nfeatures + k];
			}
		}
    }
	

#ifdef CPU_CENTER_REDUCE
#endif

#ifdef BLOCK_CENTER_REDUCE
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++)
			new_centers[j][k]= block_new_centers[j*nfeatures + k];		
	}
#endif

#endif

	return delta;
	
}


