#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>

#ifdef TIMING
#include "timing.h"
#endif

#ifdef RD_WG_SIZE_0_0
        #define MAXBLOCKSIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define MAXBLOCKSIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define MAXBLOCKSIZE RD_WG_SIZE
#else
        #define MAXBLOCKSIZE 512
#endif

#ifdef RD_WG_SIZE_1_0
        #define BLOCK_SIZE_XY RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
        #define BLOCK_SIZE_XY RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE_XY RD_WG_SIZE
#else
        #define BLOCK_SIZE_XY 4
#endif

#ifdef TIMING
struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

int Size;
float *a, *b, *finalVec;
float *m;

FILE *fp;

void InitProblemOnce(char *filename);
void InitPerRun();
void ForwardSub();
void BackSub();
__global__ void Fan1(float *m, float *a, int Size, int t);
__global__ void Fan2(float *m, float *a, float *b,int Size, int j1, int t);
void InitMat(float *ary, int nrow, int ncol);
void InitAry(float *ary, int ary_size);
void PrintMat(float *ary, int nrow, int ncolumn);
void PrintAry(float *ary, int ary_size);
void PrintDeviceProperties();
void checkCUDAError(const char *msg);

unsigned int totalKernelTime = 0;

void
create_matrix(float *m, int size){
  int i,j;
  float lamda = -0.01;
  float coe[2*size-1];
  float coe_i =0.0;

  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      coe[j]=coe_i;
      j=size-1-i;     
      coe[j]=coe_i;
    }

  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	m[i*size+j]=coe[size-1-i+j];
      }
  }
}

int main(int argc, char *argv[])
{
  printf("WG size of kernel 1 = %d, WG size of kernel 2= %d X %d\n", MAXBLOCKSIZE, BLOCK_SIZE_XY, BLOCK_SIZE_XY);
    int verbose = 0;
    int i, j;
    char flag;
    if (argc < 2) {
        printf("Usage: gaussian -f filename / -s size [-q]\n\n");
        exit(0);
    }
    
    PrintDeviceProperties();
    for(i=1;i<argc;i++) {
      if (argv[i][0]=='-') {
        flag = argv[i][1];
          switch (flag) {
            case 's':
              i++;
              Size = atoi(argv[i]);
	      printf("Create matrix internally in parse, size = %d \n", Size);
	      a = (float *) malloc(Size * Size * sizeof(float));
	      create_matrix(a, Size);
	      b = (float *) malloc(Size * sizeof(float));
	      for (j =0; j< Size; j++)
	    	b[j]=1.0;
	      m = (float *) malloc(Size * Size * sizeof(float));
              break;
            case 'f':
              i++;
	      printf("Read file from %s \n", argv[i]);
	      InitProblemOnce(argv[i]);
              break;
            case 'q':
	      verbose = 0;
              break;
	  }
      }
    }
    InitPerRun();
    struct timeval time_start;
    gettimeofday(&time_start, NULL);	
    ForwardSub();
    struct timeval time_end;
    gettimeofday(&time_end, NULL);
    unsigned int time_total = (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
    if (verbose) {
        printf("Matrix m is: \n");
        PrintMat(m, Size, Size);
        printf("Matrix a is: \n");
        PrintMat(a, Size, Size);
        printf("Array b is: \n");
        PrintAry(b, Size);
    }
    BackSub();
    if (verbose) {
        printf("The final solution is: \n");
        PrintAry(finalVec,Size);
    }
    printf("\nTime total (including memory transfers)\t%f sec\n", time_total * 1e-6);
    printf("Time for CUDA kernels:\t%f sec\n",totalKernelTime * 1e-6);
    
    free(m);
    free(a);
    free(b);

#ifdef  TIMING
	printf("Exec: %f\n", kernel_time);
#endif
}

void PrintDeviceProperties(){
	hipDeviceProp_t deviceProp;  
	int nDevCount = 0;  
	
	hipGetDeviceCount( &nDevCount );  
	printf( "Total Device found: %d", nDevCount );  
	for (int nDeviceIdx = 0; nDeviceIdx < nDevCount; ++nDeviceIdx )  
	{  
	    memset( &deviceProp, 0, sizeof(deviceProp));  
	    if( hipSuccess == hipGetDeviceProperties(&deviceProp, nDeviceIdx))  
	        {
				printf( "\nDevice Name \t\t - %s ", deviceProp.name );  
			    printf( "\n**************************************");  
			    printf( "\nTotal Global Memory\t\t\t - %lu KB", deviceProp.totalGlobalMem/1024 );  
			    printf( "\nShared memory available per block \t - %lu KB", deviceProp.sharedMemPerBlock/1024 );  
			    printf( "\nNumber of registers per thread block \t - %d", deviceProp.regsPerBlock );  
			    printf( "\nWarp size in threads \t\t\t - %d", deviceProp.warpSize );  
			    printf( "\nMemory Pitch \t\t\t\t - %zu bytes", deviceProp.memPitch );  
			    printf( "\nMaximum threads per block \t\t - %d", deviceProp.maxThreadsPerBlock );  
			    printf( "\nMaximum Thread Dimension (block) \t - %d %d %d", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2] );  
			    printf( "\nMaximum Thread Dimension (grid) \t - %d %d %d", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2] );  
			    printf( "\nTotal constant memory \t\t\t - %zu bytes", deviceProp.totalConstMem );  
			    printf( "\nCUDA ver \t\t\t\t - %d.%d", deviceProp.major, deviceProp.minor );  
			    printf( "\nClock rate \t\t\t\t - %d KHz", deviceProp.clockRate );  
			    printf( "\nTexture Alignment \t\t\t - %zu bytes", deviceProp.textureAlignment );  
			    printf( "\nDevice Overlap \t\t\t\t - %s", deviceProp. deviceOverlap?"Allowed":"Not Allowed" );  
			    printf( "\nNumber of Multi processors \t\t - %d\n\n", deviceProp.multiProcessorCount );  
			}  
	    else  
	        printf( "\n%s", hipGetErrorString(hipGetLastError()));  
	}  
}
 
 

void InitProblemOnce(char *filename)
{
	fp = fopen(filename, "r");
	
	fscanf(fp, "%d", &Size);	
	 
	a = (float *) malloc(Size * Size * sizeof(float));
	 
	InitMat(a, Size, Size);
	b = (float *) malloc(Size * sizeof(float));
	
	InitAry(b, Size);
		
	 m = (float *) malloc(Size * Size * sizeof(float));
}


void InitPerRun() 
{
	int i;
	for (i=0; i<Size*Size; i++)
			*(m+i) = 0.0;
}


__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{   
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}



__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;
	
	int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
	
	a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	if(yidx == 0){
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}


void ForwardSub()
{
	int t;
    float *m_cuda,*a_cuda,*b_cuda;
	
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(float));
	 
	hipMalloc((void **) &a_cuda, Size * Size * sizeof(float));
	
	hipMalloc((void **) &b_cuda, Size * sizeof(float));	

	hipMemcpy(m_cuda, m, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(float),hipMemcpyHostToDevice );
	
	int block_size,grid_size;
	
	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);

	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	
	int blockSize2d, gridSize2d;
	blockSize2d = BLOCK_SIZE_XY;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 
	
	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);

#ifdef  TIMING
	gettimeofday(&tv_kernel_start, NULL);
#endif

    struct timeval time_start;
    gettimeofday(&time_start, NULL);
	for (t=0; t<(Size-1); t++) {
		Fan1<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}
	struct timeval time_end;
    gettimeofday(&time_end, NULL);
    totalKernelTime = (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
	
#ifdef  TIMING
	tvsub(&time_end, &tv_kernel_start, &tv);
	kernel_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

	hipMemcpy(m, m_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(float),hipMemcpyDeviceToHost );
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
}



void BackSub()
{
	finalVec = (float *) malloc(Size * sizeof(float));
	int i,j;
	for(i=0;i<Size;i++){
		finalVec[Size-i-1]=b[Size-i-1];
		for(j=0;j<i;j++)
		{
			finalVec[Size-i-1]-=*(a+Size*(Size-i-1)+(Size-j-1)) * finalVec[Size-j-1];
		}
		finalVec[Size-i-1]=finalVec[Size-i-1]/ *(a+Size*(Size-i-1)+(Size-i-1));
	}
}

void InitMat(float *ary, int nrow, int ncol)
{
	int i, j;
	
	for (i=0; i<nrow; i++) {
		for (j=0; j<ncol; j++) {
			fscanf(fp, "%f",  ary+Size*i+j);
		}
	}  
}

void PrintMat(float *ary, int nrow, int ncol)
{
	int i, j;
	
	for (i=0; i<nrow; i++) {
		for (j=0; j<ncol; j++) {
			printf("%8.2f ", *(ary+Size*i+j));
		}
		printf("\n");
	}
	printf("\n");
}

void InitAry(float *ary, int ary_size)
{
	int i;
	
	for (i=0; i<ary_size; i++) {
		fscanf(fp, "%f",  &ary[i]);
	}
}  

void PrintAry(float *ary, int ary_size)
{
	int i;
	for (i=0; i<ary_size; i++) {
		printf("%.2f ", ary[i]);
	}
	printf("\n\n");
}
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

